#include "hip/hip_runtime.h"
﻿#include <string>
#include <iostream>
#include <fstream>
#include <vector>
#include <iomanip>
#include <random>
#include <immintrin.h>
#include <numeric>
#include <algorithm>
#include <cstdint>
#include <ranges>
#include <type_traits>
#include <cassert>

#define EIGEN_NO_CUDA
#include "Eigen/Dense"

#include ""
#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hipblaslt.h>
#include <thrust/device_ptr.h>
#include <thrust/extrema.h>
#include <hipcub/hipcub.hpp>

#include "common/cppUtil.h"
#include "common/cuUtil.h"
#include "common/simdUtil.h"
#include "common/cublasUtil.h"


// define to compare my implementation with an eigen implementation (known to work) (slow)
#undef COMPARE_MLP_WITH_EIGEN
#undef COMPARE_MLP_WITH_EIGEN_EPOCH
// compute loss function (slows down epochs)
#undef EVAL_EPOCH

/*
CUP = Cuda (multilayer) Perceptron
*/

using EigenMatrix = Eigen::Matrix<float, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor>;
using EigenRowVectorf = Eigen::RowVector<float, Eigen::Dynamic>;
using EigenVectorf = Eigen::Vector<float, Eigen::Dynamic>;
// BLOCKDIM [1 to 1024]: Number of threads per block in the CUDA kernel
constexpr size_t BLOCK_DIM = 256;


uint32_t swapEndian(uint32_t val) {
    return ((val >> 24) & 0xff) |
        ((val << 8) & 0xff0000) |
        ((val >> 8) & 0xff00) |
        ((val << 24) & 0xff000000);
}

/* a singleton class is not elegant for CublasHandle::get(), but it works for this project */
class CublasHandle {
public:
    static hipblasHandle_t& get() {
        if (mustInitCublas) {
            CUBLAS_CHECK(hipblasCreate(&cublasHandle));
            mustInitCublas = false;
        }
        return cublasHandle;
    }

    static hipblasLtHandle_t& getLt() {
        if (mustInitLt) {
            hipblasLtCreate(&ltHandle);
            mustInitLt = false;
        }
        return ltHandle;
    }

    static void free() {
        if (!mustInitCublas) {
            hipblasDestroy(cublasHandle);
        }
        if (!mustInitLt) {
            hipblasLtDestroy(ltHandle);
        }
    }
private:
    inline static bool mustInitCublas = true;
    inline static hipblasHandle_t cublasHandle;
    inline static bool mustInitLt = true;
    inline static hipblasLtHandle_t ltHandle;
};

/* host-only, owning class */
template <typename T>
class CUPRAII : public Traceable<CUPRAII<T>> {
public:
    CUPRAII(const std::vector<T>& cpuVec) : size(cpuVec.size()) {
        ptr = cuAllocCpyFromHost(cpuVec);
    }

    CUPRAII(int size) : size(size) {
        if constexpr (DEBUG) {
            std::vector<T> cpuVec(size, std::bit_cast<T>(0xBADDBADD));
            ptr = cuAllocCpyFromHost(cpuVec);
        }
        else {
            int bytes = sizeof(T) * size;
            CU_CHECK(hipMalloc(&ptr, bytes));
        }
    }

    ~CUPRAII() {
        release();
    }
    CUPRAII(const CUPRAII& rhs) : CUPRAII(rhs.size) {
        size_t bytes = sizeof(T) * rhs.size;
        CU_CHECK(hipMemcpy(ptr, rhs.ptr, bytes, hipMemcpyDeviceToDevice));
    }

    CUPRAII& operator=(const CUPRAII& rhs) {
        size_t bytes = sizeof(T) * rhs.size;
        if (this->size != rhs.size) {
            //std::cout << "CUPRAII operator= reallocating!\n";
            release();
            this->size = rhs.size;
            CU_CHECK(hipMalloc(&ptr, bytes));
        }
        CU_CHECK(hipMemcpy(ptr, rhs.ptr, bytes, hipMemcpyDeviceToDevice));
        return *this;
    }

    CUPRAII& operator=(CUPRAII&& rhs) {
        ptr = rhs.ptr;
        size = rhs.size;
        rhs.ptr = nullptr;
        rhs.size = 0;
        return *this;
    }
    CUPRAII(CUPRAII&& rhs) {
        ptr = rhs.ptr;
        size = rhs.size;
        rhs.ptr = nullptr;
        rhs.size = 0;
    }

    void release() {
        CU_CHECK(hipFree(ptr));
        ptr = nullptr;
        size = 0;
    }
    T* ptr = nullptr;
    size_t size = 0;
};

enum CUPTransMask {
    CUPNoneTrans = 0,
    CUPATrans = 1,
    CUPBTrans = 2,
    CUPABTrans = 3,
};

/* non-owning type, used in Cuda kernels */
template <typename T>
struct PODMatrix {
    int rows = 0;
    int cols = 0;
    T* data = nullptr;
    __host__ size_t getIdx(int row, int col) {
        assert(row < rows && col < cols);
        return row * cols + col;
    }
    __device__ size_t d_getIdx(int row, int col) {
        return row * cols + col;
    }

    __host__ __device__ T* end() {
        return data + rows * cols;
    }

    __host__ __device__ int size() const {
        //assert(rows * cols <= raii.size);
        return rows * cols;
    }

};

/* owning type, host only */
template <typename T>
struct CUPMatrix : public PODMatrix<T> {
    using PODMatrix<T>::rows;
    using PODMatrix<T>::cols;
    using PODMatrix<T>::data;
    using PODMatrix<T>::end;
    using PODMatrix<T>::size;
    using PODMatrix<T>::getIdx;
    CUPMatrix() = default;
    CUPMatrix(const std::vector<T>& cpuVec, int rows, int cols) : PODMatrix<T>{ rows, cols, nullptr }, raii(cpuVec) {
        if (rows * cols != cpuVec.size()) {
            throw std::runtime_error("wrong dims");
        }
        data = raii.ptr;
    }
    CUPMatrix(int rows, int cols) : PODMatrix<T>{ rows, cols, nullptr }, raii(rows* cols) {
        data = raii.ptr;
    }

    CUPMatrix(int rows, int cols, T val) : PODMatrix<T>{ rows, cols, nullptr } {
        std::vector<T> cpuVec(rows * cols, val);
        raii = CUPRAII{ cpuVec };
        data = raii.ptr;
    }

    CUPMatrix(const CUPRAII<T>& raii, int rows, int cols) : PODMatrix<T>{ rows, cols, nullptr }, raii(raii) {
        data = raii.ptr;
    }

    CUPMatrix(const CUPMatrix& rhs) : CUPMatrix(rhs.raii, rhs.rows, rhs.cols) {
        int diff = rhs.data - rhs.raii.ptr;
        data = raii.ptr + diff;
    }
    CUPMatrix& operator=(const CUPMatrix& rhs) {
        raii = rhs.raii;
        rows = rhs.rows;
        cols = rhs.cols;

        int diff = rhs.data - rhs.raii.ptr;
        data = raii.ptr + diff;
        return *this;
    }

    CUPMatrix& operator=(CUPMatrix&& rhs) = default;
    CUPMatrix(CUPMatrix&& rhs) = default;

    const PODMatrix<T> getPod() const {
        return PODMatrix<T>(*this);
    }

    static CUPMatrix Random(int rows, int cols, T minVal, T maxVal) {
        std::vector<T> ranVec(rows * cols, 0xBADDBADD);
        randSeq(ranVec.begin(), ranVec.end(), minVal, maxVal);
        return CUPMatrix(ranVec, rows, cols);
    }

    std::vector<T> cpyFromDevice() const {
        std::vector<T> cpuVec(rows * cols, 0xBADDBADD);
        cuCpyFromDevice<T>(cpuVec, data);
        return cpuVec;
    }

    void setView(size_t _rowOffset, size_t rowSpan) {
        if (_rowOffset + rowSpan > raiiRows()) {
            throw std::runtime_error("wrong view");
        }
        data = raii.ptr + _rowOffset * cols;
        rows = rowSpan;
        assert(data >= raii.ptr && end() <= raii.ptr + raii.size);
    }

    int getRowOffset() {
        int diff = data - raii.ptr;
        assert(diff % cols == 0);
        return diff / cols;
    }

    const CUPRAII<T>& getRaii() const {
        return raii;
    }

    int raiiRows() const {
        assert(raii.size % cols == 0);
        return raii.size / cols;
    }

    template <CUPTransMask transMask = CUPNoneTrans>
    void gemm(const CUPMatrix<T>& aMatrix, const CUPMatrix<T>& bMatrix, float alpha = 1.f, float beta = 0.f) {

        const float* A = aMatrix.data;
        const float* B = bMatrix.data;
        int M, N, K;
        int lda, ldb;

        hipblasOperation_t aTransOpt;
        hipblasOperation_t bTransOpt;

        if constexpr (transMask == CUPNoneTrans) {
            if (aMatrix.cols != bMatrix.rows) {
                throw std::runtime_error("wrong dim");
            }
            M = aMatrix.rows;
            K = aMatrix.cols;
            N = bMatrix.cols;
            lda = K;
            ldb = N;
            aTransOpt = HIPBLAS_OP_N;
            bTransOpt = HIPBLAS_OP_N;
        }
        else if constexpr (transMask == CUPATrans) {
            if (aMatrix.rows != bMatrix.rows) {
                throw std::runtime_error("wrong dim");
            }
            M = aMatrix.cols; // A trans!
            K = aMatrix.rows; // A trans!
            N = bMatrix.cols;
            lda = M;
            ldb = N;
            aTransOpt = HIPBLAS_OP_T;
            bTransOpt = HIPBLAS_OP_N;
        }
        else if constexpr (transMask == CUPBTrans) {
            if (aMatrix.cols != bMatrix.cols) {
                throw std::runtime_error("wrong dim");
            }
            M = aMatrix.rows;
            K = aMatrix.cols;
            N = bMatrix.rows; // B trans!
            lda = K;
            ldb = K;
            aTransOpt = HIPBLAS_OP_N;
            bTransOpt = HIPBLAS_OP_T;
        }
        else if constexpr (transMask == (CUPATrans | CUPBTrans)) {
            if (aMatrix.rows != bMatrix.cols) {
                throw std::runtime_error("wrong dim");
            }
            M = aMatrix.cols; // A trans!
            K = aMatrix.rows; // A trans!
            N = bMatrix.rows; // B trans!
            lda = M;
            ldb = K;
            aTransOpt = HIPBLAS_OP_T;
            bTransOpt = HIPBLAS_OP_T;
        }
        else {
            static_assert(false, "wrong mask");
        }

        if (M != rows || N != cols) {
            raii.release();
            std::cout << "CUPMatrix<T>::gemm reallocate\n";
            *this = CUPMatrix<T>{ M, N };
        }
        float* C = data;

        //std::cout << "aTransOpt: " << aTransOpt << "\n"
        //    << "bTransOpt: " << bTransOpt << "\n"
        //    << "M: " << M << "\n"
        //    << "N: " << N << "\n"
        //    << "K: " << K << "\n"
        //    << "alpha: " << alpha << "\n"
        //    << "lda: " << lda << "\n"
        //    << "ldb: " << ldb << "\n"
        //    << "beta: " << beta << "\n"
        //    << "N: " << N << "\n";

        int ldc = N;
        size_t workspaceSize = 0;
        void* workspace = nullptr; // or allocate a workspace if desired
        LtSgemm(CublasHandle::getLt(),
            aTransOpt,
            bTransOpt,
            M,
            N,
            K,
            &alpha, /* host pointer */
            A,
            lda,
            B,
            ldb,
            &beta, /* host pointer */
            C,
            ldc,
            workspace,
            workspaceSize);
    }

    void colwiseSumAlpha(const CUPMatrix<T>& mat, CUPMatrix<T>& ones, float alpha, float beta = 0.f) {
        // resize `ones` if necessary:
        if (ones.cols != 1 || ones.rows < mat.rows) {
            std::cout << "ColwiseSumAlpha reallocate ones\n";
            ones = CUPMatrix<T>(mat.rows, 1, 1.f);
        }
        if (this->rows != 1 || this->cols != mat.cols) {
            std::cout << "ColwiseSumAlpha reallocate C\n";
            *this = CUPMatrix<T>(1, mat.cols);
        }

        int N = mat.rows;
        int M = mat.cols;

        hipblasSgemv(CublasHandle::get(),
            HIPBLAS_OP_N, // transpose A
            M, N,        // dimensions of A
            &alpha,
            mat.data, M,   // A pointer and leading dimension
            ones.data, 1,       // ones vector
            &beta,
            this->data, 1);       // output vector

    }

    void positiveMask(const CUPMatrix<T>& mask) {
        assert(cols == mask.cols && rows == mask.rows);
        int blocks = (size() + BLOCK_DIM - 1) / BLOCK_DIM;
        cuPositiveMask << <blocks, BLOCK_DIM >> > (getPod(), mask.getPod());
        hipDeviceSynchronize();
    }

    //void dup_rows(const CUPMatrix<T>& row, int numRows) {
    //    assert(row.cols == 1); // we are assuming a row vector here!
    //    // O(log n) memcpy calls
    //    if (rows != numRows || cols != row.cols) {
    //        raii.release();
    //        *this = CUPMatrix<T>{ numRows, row.rows };
    //    }

    //    if (row.size() != cols) {
    //        int _rowSize = row.size();
    //        throw std::runtime_error("wrong size " + std::to_string(_rowSize));
    //    }

    //    size_t rowSize = sizeof(T) * row.size();
    //    CU_CHECK(hipMemcpy(data, row.data, rowSize, hipMemcpyDeviceToDevice));

    //    size_t copied = 1;
    //    char* d = reinterpret_cast<char*>(data);

    //    // double copy region every time
    //    while (copied < rows) {
    //        size_t rowsToCopy = std::min(copied, rows - copied);
    //        CU_CHECK(hipMemcpy(d + copied * rowSize, d, rowsToCopy * rowSize, hipMemcpyDeviceToDevice));
    //        copied += rowsToCopy;
    //    }
    //}
    void dupRows2(const CUPMatrix<T>& row, int numRows) {
        assert(row.cols == 1); // we are assuming a row vector here!
        // O(log n) memcpy calls
        if (rows != numRows || cols != row.rows) {
            raii.release();
            *this = CUPMatrix<T>{ numRows, row.rows };
        }

        if (row.size() != cols) {
            int _rowSize = row.size();
            throw std::runtime_error("wrong size " + std::to_string(_rowSize));
        }

        int blocks = (size() + BLOCK_DIM - 1) / BLOCK_DIM;
        dim3 blockDim(32, 8);
        dim3 gridDim((cols + blockDim.x - 1) / blockDim.x,
            (rows + blockDim.y - 1) / blockDim.y);

        cuDupRows2 << <gridDim, blockDim >> > (getPod(), row.getPod());
        hipDeviceSynchronize();

    }


    void relu() {
        int blocks = (size() + BLOCK_DIM - 1) / BLOCK_DIM;
        cuRelu << <blocks, BLOCK_DIM >> > (getPod());
        hipDeviceSynchronize();
    }

    void oneHot(const CUPMatrix<int>& y, int maxVal) {
        assert(y.cols == 1);
        if (raii.size != y.rows * maxVal) {
            *this = CUPMatrix(y.rows, maxVal);
        }
        rows = y.rows;
        cols = maxVal;

        int blocks = (size() + BLOCK_DIM - 1) / BLOCK_DIM;
        cuOneHot << <blocks, BLOCK_DIM >> > (getPod(), y.getPod());
        hipDeviceSynchronize();

    }

    void softmax() requires std::same_as<T, float> {
        // matrix.cols is currently hardcoded for this function. TODO unhardcode!
        constexpr int BlockSize = 32; // must be multiple of warp size
        if (cols > BlockSize) {
            throw std::runtime_error("unhardcode me");
        }

        int blocks = rows;
        cuSoftmax<BlockSize> << <blocks, BlockSize >> > (getPod());
        hipDeviceSynchronize();
    }

    CUPRAII<T> raii{ 0 };
};

//BlockSize given as template because we need it to be constexpr
template <int BlockSize>
__global__ void cuSoftmax(PODMatrix<float> mat) {
    assert(blockIdx.x < mat.rows);
    static_assert(BlockSize % 32 == 0); // BlockReduce requires multiple of warp size
    // Each block processes one row
    using BlockReduce = hipcub::BlockReduce<float, BlockSize>;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    __shared__ float sync_reduce;

    int idx = mat.d_getIdx(blockIdx.x, threadIdx.x);
    float thread_max = (threadIdx.x < mat.cols) ? mat.data[idx] : mat.data[0];

    // Only thread_0 in block has the correct result
    float block_max = BlockReduce(temp_storage).Reduce(thread_max, hipcub::Max());
    //v sync block
    if (threadIdx.x == 0) {
        sync_reduce = block_max;
    }
    __syncthreads();
    block_max = sync_reduce;
    //^ sync block

    if (threadIdx.x < mat.cols) {
        mat.data[idx] -= block_max;
        mat.data[idx] = expf(mat.data[idx]);
    }

    float thread_sum = (threadIdx.x < mat.cols) ? mat.data[idx] : 0;
    __syncthreads(); // required because of temp_storage reuse
    // Only thread_0 in block has the correct result
    float block_sum = BlockReduce(temp_storage).Reduce(thread_sum, hipcub::Sum());
    //v sync block
    if (threadIdx.x == 0) {
        sync_reduce = block_sum;
    }
    __syncthreads();
    block_sum = sync_reduce;
    //^ sync block

    if (threadIdx.x < mat.cols) {
        mat.data[idx] /= block_sum;
    }
}

__global__ void cuOneHot(PODMatrix<float> lhs, const PODMatrix<int> y) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int row = idx / lhs.cols;
    int col = idx % lhs.cols;
    if (idx < lhs.size()) {
        lhs.data[idx] = 0;
        if (col == y.data[row]) {
            lhs.data[idx] = 1.0f;
        }
    }
}

__global__ void cuRelu(PODMatrix<float> mat) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < mat.size())
        mat.data[idx] = mat.data[idx] > 0 ? mat.data[idx] : 0;
}

__global__ void cuDupRows2(PODMatrix<float> dst, PODMatrix<float> srcRow) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure we're within bounds
    if (col < dst.cols && row < dst.rows) {
        dst.data[row * dst.cols + col] = srcRow.data[col];
    }
}

__global__ void cuPositiveMask(PODMatrix<float> mat, const PODMatrix<float> mask) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < mat.size()) {
        if (mask.data[idx] <= 0) {
            mat.data[idx] = 0;
        }
    }

}


template <typename T>
CUPMatrix<T> readIdxXubyte(const std::string& dataFile) {
    // T == float: we are reading data
    // T == int: we are reading labels
    std::cout << "loading " << dataFile << std::endl;
    std::ifstream dataIfstream(dataFile, std::ios::binary);
    if (!dataIfstream) {
        std::cerr << "Unable to open file: " << dataFile << std::endl;
        exit(-1);
    }

    // Read header: magic number, number of images, rows, and columns.
    uint32_t magicNumber = 0;
    dataIfstream.read(reinterpret_cast<char*>(&magicNumber), sizeof(magicNumber));
    magicNumber = swapEndian(magicNumber);
    std::cout << "Magic Number: " << magicNumber << "\n";

    constexpr int numDim = std::is_same_v<T, int> ? 1 : 3;
    std::array<uint32_t, numDim> dim;
    for (int i = 0; i < numDim; ++i) {
        dataIfstream.read(reinterpret_cast<char*>(&(dim[i])), sizeof(uint32_t));
        dim[i] = swapEndian(dim[i]);
        std::cout << "Dim" << i << ": " << dim[i] << "\n";
    }

    // Read data:
    int totalElements = std::accumulate(dim.cbegin(), dim.cend(), 1, [](int a, const int& b) {return a * b; });
    std::vector<T> cpuData(totalElements, 0xBADDBADD);
    for (int i = 0; i < cpuData.size(); ++i) {
        uint8_t byte;
        dataIfstream.read(reinterpret_cast<char*>(&byte), sizeof(byte));
        if (!dataIfstream) {
            throw std::runtime_error("error reading pos " + i);
        }


        cpuData[i] = T(byte);
        if constexpr (std::is_same_v<T, float>) {
            cpuData[i] /= 255.f;
        }
    }

    // lambda for guaranteed copy elision
    CUPMatrix<T> mat = [&] {
        if constexpr (std::is_same_v<T, int>)
            return CUPMatrix<T>{ cpuData, int(dim[0]), 1 };
        else
            return CUPMatrix<T>{ cpuData, int(dim[0]), int(dim[1] * dim[2]) };
        }();


    return mat;
}

namespace _testEig {
    // EigenType: EigenMatrix or EigenRowVectorF
    template <typename T>
    EigenMatrix fromCUPMatrix(const CUPMatrix<T>& cup);
    EigenRowVectorf fromCUPVector(const CUPMatrix<float>& cup);
    EigenMatrix relu(const EigenMatrix& x);
    EigenMatrix softmax(const EigenMatrix& x);
    EigenMatrix one_hot(const EigenMatrix& y, int maxVal);
    EigenMatrix dup_rows(const EigenMatrix& x, const EigenRowVectorf& y);
    EigenMatrix positive_mask(const EigenMatrix& data, const EigenMatrix& mask);
    void cmpMat(const EigenMatrix& a, const EigenMatrix& b);
}

class MLP {

public:
    MLP(CUPMatrix<float>& x, CUPMatrix<int>& y, size_t hiddenSize, int batchSize, float lr, int epochs) {

        size_t inputSize = x.cols;
        auto d_begin = thrust::device_pointer_cast(y.data);
        auto d_end = thrust::device_pointer_cast(y.end());
        auto max_iter = thrust::max_element(d_begin, d_end);
        size_t outputSize = *max_iter + 1;

        std::vector<float>initVec(inputSize * hiddenSize, 0xBADDBADD);
        randSeq(initVec.begin(), initVec.end(), -0.01f, 0.01f);
        weight_1 = CUPMatrix<float>(initVec, inputSize, hiddenSize);

        initVec = std::vector<float>(hiddenSize, 0xBADDBADD);
        randSeq(initVec.begin(), initVec.end(), 0.f, 1.f);
        bias_1 = CUPMatrix<float>(initVec, hiddenSize, 1);

        initVec = std::vector<float>(hiddenSize * outputSize, 0xBADDBADD);
        randSeq(initVec.begin(), initVec.end(), -0.01f, 0.01f);
        weight_2 = CUPMatrix<float>(initVec, hiddenSize, outputSize);

        initVec = std::vector<float>(outputSize, 0xBADDBADD);
        randSeq(initVec.begin(), initVec.end(), 0.f, 1.f);
        bias_2 = CUPMatrix<float>(initVec, outputSize, 1);

        std::cout << "Epoch\tLoss\n";
        for (int epoch = 0; epoch < epochs; ++epoch) {

            Time begin = getTime();
            for (int i = 0; i < x.raiiRows(); i += x.rows) {
                int batchRows = std::min((x.raiiRows() - i), batchSize);
                x.setView(i, batchRows);
                y.setView(i, batchRows);
                forward(x);
                backward(x, y, lr);
            }
            Seconds elapsed = getTime() - begin;
            std::cout << "epoch time (" << epoch << "/" << epochs << "): " << elapsed << std::endl;

#ifdef EVAL_EPOCH
            x.setView(0, x.raiiRows());
            y.setView(0, y.raiiRows());
            evalEpoch(x, y, epoch);
#endif
        }
    }


    /*
    input: all training data
    startRow: this batch begins at startRow
    batchRows: this batch ends at endRow = startRow + batchSize
    */
    void forward(const CUPMatrix<float>& x) {
#ifdef COMPARE_MLP_WITH_EIGEN
        EigenMatrix batchEig = _testEig::fromCUPMatrix<float>(x);
        EigenMatrix weight_1Eig = _testEig::fromCUPMatrix<float>(weight_1);
        EigenRowVectorf bias_1Eig = _testEig::fromCUPVector(bias_1);
        EigenMatrix z1Eig = (batchEig * weight_1Eig).rowwise() + bias_1Eig;

        EigenMatrix a1Eig = _testEig::relu(z1Eig);

        EigenMatrix weight_2Eig = _testEig::fromCUPMatrix<float>(weight_2);
        EigenRowVectorf bias_2Eig = _testEig::fromCUPVector(bias_2);
        EigenMatrix z2Eig = (a1Eig * weight_2Eig).rowwise() + bias_2Eig;

        EigenMatrix a2Eig = _testEig::softmax(z2Eig);
#endif // COMPARE_MLP_WITH_EIGEN
        //z1.dup_rows(bias_1, x.rows);
        z1.dupRows2(bias_1, x.rows);
        z1.gemm(x, weight_1, 1.f, 1.f);

        a1 = z1;
        a1.relu();

        //z2 = (a1 * weight_2) +(rowWise) bias_2;
        //z2.dup_rows(bias_2, x.rows);
        z2.dupRows2(bias_2, x.rows);
        z2.gemm(a1, weight_2, 1.f, 1.f);

        a2 = z2;
        a2.softmax();

#ifdef COMPARE_MLP_WITH_EIGEN
        EigenMatrix z1Cmp = _testEig::fromCUPMatrix<float>(z1);
        EigenMatrix a1Cmp = _testEig::fromCUPMatrix<float>(a1);
        EigenMatrix z2Cmp = _testEig::fromCUPMatrix<float>(z2);
        EigenMatrix a2Cmp = _testEig::fromCUPMatrix<float>(a2);

        _testEig::cmpMat(z1Cmp, z1Eig);
        _testEig::cmpMat(a1Cmp, a1Eig);
        _testEig::cmpMat(z2Cmp, z2Eig);
        _testEig::cmpMat(a2Cmp, a2Eig);
#endif // COMPARE_MLP_WITH_EIGEN
    }
    /*
    input: all training data
    y: all labels
    startRow: this batch begins at startRow
    batchRows: this batch ends at endRow = startRow + batchSize
    */
    void backward(const CUPMatrix<float>& batchX, const CUPMatrix<int>& batchY, float lr) {
        float divM = 1.f / batchX.rows;
#ifdef COMPARE_MLP_WITH_EIGEN
        EigenMatrix batchYEig = _testEig::fromCUPMatrix<int>(batchY); // note int->float
        EigenMatrix y_one_hotEig = _testEig::one_hot(batchYEig, a2.cols); // a2.cols = outputSize

        EigenMatrix a2Eig = _testEig::fromCUPMatrix<float>(a2);
        EigenMatrix dL_dz2Eig = a2Eig - y_one_hotEig;

        EigenMatrix a1Eig = _testEig::fromCUPMatrix<float>(a1);
        EigenMatrix dL_dW2Eig = (a1Eig.transpose() * dL_dz2Eig) * divM;

        EigenMatrix weight_2Eig = _testEig::fromCUPMatrix<float>(weight_2);
        weight_2Eig -= lr * dL_dW2Eig;

        EigenMatrix dL_db2Eig = (dL_dz2Eig.colwise().sum() * divM);

        EigenMatrix bias_2Eig = _testEig::fromCUPMatrix<float>(bias_2);
        bias_2Eig -= lr * dL_db2Eig.transpose();

        EigenMatrix dL_da1Eig = dL_dz2Eig * weight_2Eig.transpose();

        EigenMatrix z1Eig = _testEig::fromCUPMatrix<float>(z1);
        EigenMatrix dL_dz1Eig = (dL_da1Eig.array() * (z1Eig.array() > 0).cast<float>()).matrix();

        EigenMatrix batchXEig = _testEig::fromCUPMatrix<float>(batchX);
        EigenMatrix dL_dW1Eig = (batchXEig.transpose() * dL_dz1Eig) * divM;

        EigenMatrix weight_1Eig = _testEig::fromCUPMatrix<float>(weight_1);
        weight_1Eig -= lr * dL_dW1Eig;

        EigenMatrix dL_db1Eig = (dL_dz1Eig.colwise().sum() * divM);

        EigenMatrix bias_1Eig = _testEig::fromCUPMatrix<float>(bias_1);
        bias_1Eig -= lr * dL_db1Eig.transpose();
#endif // COMPARE_MLP_WITH_EIGEN
        int outputSize = a2.cols;
        y_one_hot.oneHot(batchY, outputSize);

        // 2. Compute gradient at output layer:
        // dL_dz2 = a2 - y_one_hot;
        dL_dz2 = a2;
        float alpha = -1.f;
        CUBLAS_CHECK(
            hipblasSaxpy(     // y = y + alpha * x
                CublasHandle::get(),
                dL_dz2.size(), //n
                &alpha,		 //alpha
                y_one_hot.data, //x
                1,			//incx
                dL_dz2.data,    //y
                1			//incy
            )
        );

        // 3. Gradients for the second (output) layer:
        dL_dW2.gemm<CUPATrans>(a1, dL_dz2, divM);

        // weight_2 -= lr * dL_dW2
        alpha = -lr;

        CUBLAS_CHECK(
            hipblasSaxpy( // y = y + alpha * x
                CublasHandle::get(),
                dL_dW2.size(), // x.size
                &alpha,			 // alpha
                dL_dW2.data,     // x
                1,				 // incx
                weight_2.data,// y
                1 // incy
            )
        );

        // dL_db2 = dL_dz2.colwise().sum() * divM
        dL_db2.colwiseSumAlpha(dL_dz2, ones, divM);

        // bias_2 -= lr * dL_db2
        alpha = -lr;
        CUBLAS_CHECK(
            hipblasSaxpy(		 // y = y + alpha * x
                CublasHandle::get(),
                dL_db2.size(), // x.size
                &alpha,			 // alpha
                dL_db2.data, // x
                1,				 // incx
                bias_2.data, // y
                1 				 // incy
            )
        );

        // 4. Backpropagate to the hidden layer:
        dL_da1.gemm<CUPBTrans>(dL_dz2, weight_2);

        dL_dz1 = dL_da1;
        dL_dz1.positiveMask(z1);

        // 5. Gradients for the first (hidden) layer:
        dL_dW1.gemm<CUPATrans>(batchX, dL_dz1, divM);

        // weight_1 -= lr * dL_dW1
        alpha = -lr;
        CUBLAS_CHECK(
            hipblasSaxpy( // y = y + alpha * x
                CublasHandle::get(),
                dL_dW1.size(), // x.size
                &alpha,			 // alpha
                dL_dW1.data,// x
                1,				 // incx
                weight_1.data, // y
                1 				 // incy
            )
        );


        // dL_db1 = dL_dz1.colwise().sum() * divM
        dL_db1.colwiseSumAlpha(dL_dz1, ones, divM);

        //bias_1 -= lr * dL_db1
        alpha = -lr;
        CUBLAS_CHECK(
            hipblasSaxpy(		 // y = y + alpha * x
                CublasHandle::get(),
                dL_db1.size(), // x.size
                &alpha,			 // alpha
                dL_db1.data, // x
                1,				 // incx
                bias_1.data, // y
                1 				 // incy
            )
        );

#ifdef COMPARE_MLP_WITH_EIGEN
        EigenMatrix y_one_hotCmp = _testEig::fromCUPMatrix<float>(y_one_hot);
        _testEig::cmpMat(y_one_hotCmp, y_one_hotEig);
        EigenMatrix dL_dz2Cmp = _testEig::fromCUPMatrix<float>(dL_dz2);
        _testEig::cmpMat(dL_dz2Cmp, dL_dz2Eig);
        EigenMatrix dL_dW2Cmp = _testEig::fromCUPMatrix<float>(dL_dW2);
        _testEig::cmpMat(dL_dW2Cmp, dL_dW2Eig);
        EigenMatrix weight_2Cmp = _testEig::fromCUPMatrix<float>(weight_2);
        _testEig::cmpMat(weight_2Cmp, weight_2Eig);
        EigenMatrix dL_db2Cmp = _testEig::fromCUPMatrix<float>(dL_db2);
        _testEig::cmpMat(dL_db2Cmp, dL_db2Eig);
        EigenMatrix bias_2Cmp = _testEig::fromCUPMatrix<float>(bias_2);
        _testEig::cmpMat(bias_2Cmp, bias_2Eig);
        EigenMatrix dL_da1Cmp = _testEig::fromCUPMatrix<float>(dL_da1);
        _testEig::cmpMat(dL_da1Cmp, dL_da1Eig);
        EigenMatrix dL_dz1Cmp = _testEig::fromCUPMatrix<float>(dL_dz1);
        _testEig::cmpMat(dL_dz1Cmp, dL_dz1Eig);
        EigenMatrix dL_dW1Cmp = _testEig::fromCUPMatrix<float>(dL_dW1);
        _testEig::cmpMat(dL_dW1Cmp, dL_dW1Eig);
        EigenMatrix weight_1Cmp = _testEig::fromCUPMatrix<float>(weight_1);
        _testEig::cmpMat(weight_1Cmp, weight_1Eig);
        EigenMatrix dL_db1Cmp = _testEig::fromCUPMatrix<float>(dL_db1);
        _testEig::cmpMat(dL_db1Cmp, dL_db1Eig);
        EigenMatrix bias_1Cmp = _testEig::fromCUPMatrix<float>(bias_1);
        _testEig::cmpMat(bias_1Cmp, bias_1Eig);
#endif
    }

    void evalEpoch(const CUPMatrix<float>& x, const CUPMatrix<int>& y, int epoch) {
        std::cout << "evalEpoch not implemented\n";
        float epsilon = 1.0e-6F;
        forward(x); // output is in `a2` member var

        //__m256 epoch_loss = _mm256_setzero_ps();
        //size_t outSize = a2.cols;
        //int batchSize = a2.rows;
        //for (int row = 0; row < batchSize; row += 8) {
        //    __m256 probs = _mm256_set_ps(
        //        *a2.data32(row, y.at32(y.gemmOffset + row)),
        //        *a2.data32(row + 1, y.at32(y.gemmOffset + row + 1)),
        //        *a2.data32(row + 2, y.at32(y.gemmOffset + row + 2)),
        //        *a2.data32(row + 3, y.at32(y.gemmOffset + row + 3)),
        //        *a2.data32(row + 4, y.at32(y.gemmOffset + row + 4)),
        //        *a2.data32(row + 5, y.at32(y.gemmOffset + row + 5)),
        //        *a2.data32(row + 6, y.at32(y.gemmOffset + row + 6)),
        //        *a2.data32(row + 7, y.at32(y.gemmOffset + row + 7)));
        //    probs = _mm256_add_ps(probs, _mm256_set1_ps(epsilon)); // against log(0)
        //    probs = _mm256_log_ps(probs);
        //    epoch_loss = _mm256_add_ps(epoch_loss, probs);
        //}

        //float epoch_loss_sum = sum256f(epoch_loss);
        //epoch_loss_sum /= -int(batchSize);
        //losses.push_back(epoch_loss_sum);

        //std::cout << (epoch + 1) << "\t" << epoch_loss_sum << std::endl;



#ifdef COMPARE_MLP_WITH_EIGEN_EPOCH
        auto a2ptr = thrust::device_pointer_cast(a2.data);
        EigenMatrix yEig = _testEig::fromCUPMatrix<int>(y);
        assert(yEig.cols() == 1);
        double epoch_lossEig = 0.0;
        for (size_t row = 0; row < a2.rows; ++row) {
            auto elptr = a2ptr + row * a2.cols + yEig(row, 0);
            assert(elptr.get() < a2.end());
            float prob = *elptr;
            prob += epsilon; // against log(0)
            epoch_lossEig += std::log(prob);
        }
        epoch_lossEig /= -a2.rows;
        std::cout << (epoch + 1) << "\t" << epoch_lossEig << std::endl;

        //float lastLoss = losses.back();
        //if (std::fabsf(epoch_lossEig - lastLoss) > 0.001f) {
        //    throw std::runtime_error("wrong loss");
        //}
#endif // COMPARE_MLP_WITH_EIGEN

    }

    std::vector<int> predict(const CUPMatrix<float>& testX) {
        forward(testX);

        std::vector<int> predictions(testX.rows, 0xBADDBADD);
        EigenMatrix a2Eig = _testEig::fromCUPMatrix<float>(a2);
        for (int i = 0; i < a2Eig.rows(); ++i) {
            int maxIndex;
            a2Eig.row(i).maxCoeff(&maxIndex);
            predictions[i] = maxIndex;
        }

        return predictions;
    }

    CUPMatrix<float> weight_1; //< dim [inputSize x hiddenSize]
    CUPMatrix<float> bias_1;   //< dim [1 x hiddenSize]
    CUPMatrix<float> weight_2; //< dim [hiddenSize x outputSize]
    CUPMatrix<float> bias_2;   //< dim [1 x outputSize]

    CUPMatrix<float> z1; //< dim [batchSize x hiddenSize]
    CUPMatrix<float> z2; //< dim [batchSize x outputSize]
    CUPMatrix<float> a1; //< dim [batchSize x hiddenSize]
    CUPMatrix<float> a2; //< dim [batchSize x outputSize]

    // temp matrices
    CUPMatrix<float> y_one_hot;
    CUPMatrix<float> dL_dz2;
    CUPMatrix<float> dL_dW2; // dim [hiddenSize x outputSize]
    CUPMatrix<float> dL_db2;
    CUPMatrix<float> dL_da1;
    CUPMatrix<float> dL_dz1;
    CUPMatrix<float> dL_dW1;
    CUPMatrix<float> dL_db1;
    CUPMatrix<float> ones;

    std::vector<float> losses;
};

/* Eigen implementations for comparisons with my implementations */
namespace _testEig {
    void printEig(const EigenMatrix& mat, const std::string& name) {
        int printPrecision = 3;
        Eigen::IOFormat fmt(printPrecision, 0, ", ", "\n", "[", "]"); // up printPrecision if diff hard to spot
        std::cerr << name << std::endl << mat.format(fmt) << std::endl;
    }

    using ::EigenMatrix;
    template <typename T>
    EigenMatrix fromCUPMatrix(const CUPMatrix<T>& cup) {
        std::vector<T> cpuVec = cup.cpyFromDevice();
        EigenMatrix eig = EigenMatrix(cup.rows, cup.cols);
        for (int row = 0; row < cup.rows; ++row) {
            for (int col = 0; col < cup.cols; ++col) {
                eig(row, col) = float(cpuVec[row * cup.cols + col]);
            }

        }
        return eig;
    }

    EigenRowVectorf fromCUPVector(const CUPMatrix<float>& cup) {
        std::vector<float> cpuVec = cup.cpyFromDevice();
        EigenMatrix eig = EigenRowVectorf(cup.rows);
        for (int row = 0; row < cup.rows; ++row) {
            eig(row) = cpuVec[row];
        }
        return eig;
    }

    EigenMatrix relu(const EigenMatrix& x) {
        return x.cwiseMax(0.0);
    }

    EigenMatrix softmax(const EigenMatrix& x) {
        EigenMatrix rowMax = x.rowwise().maxCoeff();
        EigenMatrix x_stable = x - rowMax.replicate(1, x.cols());
        EigenMatrix exp_x = x_stable.array().exp();
        EigenVectorf rowSum = exp_x.rowwise().sum();
        EigenMatrix sm = exp_x.array().colwise() / rowSum.array();
        return sm;
    }

    EigenMatrix one_hot(const EigenMatrix& y, int maxVal) {
        assert(y.cols() == 1);
        EigenMatrix y_one_hot = EigenMatrix(y.rows(), maxVal);
        y_one_hot.setZero();
        for (int i = 0; i < y_one_hot.rows(); ++i) {
            int label = y(i);
            y_one_hot(i, label) = 1.0f;
        }
        return y_one_hot;
    }

    EigenMatrix dup_rows(const EigenMatrix& x, const EigenMatrix& y) {
        assert(x.cols() == y.rows() && y.cols() == 1); // we are assuming a rowvector here!
        return x.rowwise() + EigenRowVectorf(y.transpose());
    }

    EigenMatrix positive_mask(const EigenMatrix& data, const EigenMatrix& mask) {
        return (data.array() * (mask.array() > 0).cast<float>()).matrix();
    }

    void cmpMat(const EigenMatrix& a, const EigenMatrix& b) {
        if (a.isApprox(b)) {
            //std::cout << "Test passed: matrices are equal." << std::endl;
        }
        else {
            std::cerr << "Test failed: matrices differ." << std::endl;
            printEig(a, "Matrix A");
            printEig(b, "Matrix B");
            assert(false);
        }
    }

    void statistics(EigenMatrix& mat) {
        float minVal = mat.minCoeff();
        float maxVal = mat.maxCoeff();
        float sumVal = mat.sum();

        std::cout << "Data: min = " << minVal
            << ", max = " << maxVal
            << ", sum = " << sumVal << std::endl;

    }

    void printImage(const EigenMatrix& mat, int imageId, int rows, int cols) {
        for (size_t y = 0; y < rows; ++y) {
            for (size_t x = 0; x < cols; ++x) {
                float val = mat(imageId, y * cols + x);
                std::cout << ASCIIArtFromFloat(val);
            }
            std::cout << std::endl;
        }
    }

}

//// ^TESTEIGEN
///////////////////////////////////////////////////

bool nextPermute(std::vector<int>& in, std::vector<int>& out) {

    int n = in.size();
    int k = out.size();
    for (int i = 0; i < k; i++)
    {
        out[i] = in[i];
    }
    std::reverse(in.begin() + k, in.end());
    return std::next_permutation(in.begin(), in.end());
}

void test_gemm(int m, int n, int k) {
    // Test 1: MlpNoneTrans with multi-dim aMatrix
    {
        //std::cout << m << " " << n << " " << k << "\n";
        CUPMatrix<float> mata = CUPMatrix<float>::Random(m, k * 8, -5.f, 5.f);
        CUPMatrix<float> matb = CUPMatrix<float>::Random(k * 8, n * 8, -5.f, 5.f);
        CUPMatrix<float> matc = CUPMatrix<float>::Random(m, n * 8, -5.f, 5.f);
        matc.gemm<CUPNoneTrans>(mata, matb, 1.f);

        EigenMatrix eigA = _testEig::fromCUPMatrix<float>(mata);
        EigenMatrix eigB = _testEig::fromCUPMatrix<float>(matb);
        EigenMatrix eigCmp = eigA * eigB;
        EigenMatrix mlpCmp = _testEig::fromCUPMatrix<float>(matc);

        _testEig::cmpMat(eigCmp, mlpCmp);
    }


    // Test 3: MlpATrans
    {
        CUPMatrix<float> mata = CUPMatrix<float>::Random(k, m * 8, -5.f, 5.f);
        CUPMatrix<float> matb = CUPMatrix<float>::Random(k, n * 8, -5.f, 5.f);
        CUPMatrix<float> matc = CUPMatrix<float>::Random(m * 8, n * 8, -5.f, 5.f);
        matc.gemm<CUPATrans>(mata, matb, 1.f);

        EigenMatrix eigA = _testEig::fromCUPMatrix<float>(mata).transpose();
        EigenMatrix eigB = _testEig::fromCUPMatrix<float>(matb);
        EigenMatrix eigCmp = eigA * eigB;
        EigenMatrix mlpCmp = _testEig::fromCUPMatrix<float>(matc);

        _testEig::cmpMat(eigCmp, mlpCmp);
    }

    // Test 5: MlpBTrans with multi-dim aMatrix
    {
        CUPMatrix<float> mata = CUPMatrix<float>::Random(m, k * 8, -5.f, 5.f);
        CUPMatrix<float> matb = CUPMatrix<float>::Random(n * 8, k * 8, -5.f, 5.f);
        CUPMatrix<float> matc = CUPMatrix<float>::Random(m, n * 8, -5.f, 5.f);
        matc.gemm<CUPBTrans>(mata, matb, 1.f);

        EigenMatrix eigA = _testEig::fromCUPMatrix<float>(mata);
        EigenMatrix eigB = _testEig::fromCUPMatrix<float>(matb).transpose();
        EigenMatrix eigCmp = eigA * eigB;
        EigenMatrix mlpCmp = _testEig::fromCUPMatrix<float>(matc);

        _testEig::cmpMat(eigCmp, mlpCmp);
    }

    // Test 7: (MlpATrans | MlpBTrans) with multi-dim aMatrix
    {
        CUPMatrix<float> mata = CUPMatrix<float>::Random(k * 8, m * 8, -5.f, 5.f);
        CUPMatrix<float> matb = CUPMatrix<float>::Random(n * 8, k * 8, -5.f, 5.f);
        CUPMatrix<float> matc = CUPMatrix<float>::Random(m * 8, n * 8, -5.f, 5.f);
        matc.gemm<CUPABTrans>(mata, matb, 1.f);

        EigenMatrix eigA = _testEig::fromCUPMatrix<float>(mata).transpose();
        EigenMatrix eigB = _testEig::fromCUPMatrix<float>(matb).transpose();
        EigenMatrix eigCmp = eigA * eigB;
        EigenMatrix mlpCmp = _testEig::fromCUPMatrix<float>(matc);

        _testEig::cmpMat(eigCmp, mlpCmp);
    }
}

void test_relu(int m, int n) {
    n *= 8;
    CUPMatrix<float> mlp = CUPMatrix<float>::Random(m, n * 8, -5.f, 5.f);
    EigenMatrix eig = _testEig::fromCUPMatrix<float>(mlp);
    mlp.relu();
    EigenMatrix mlpCmp = _testEig::fromCUPMatrix<float>(mlp);
    EigenMatrix eigCmp = _testEig::relu(eig);

    _testEig::cmpMat(eigCmp, mlpCmp);
}
void test_softmax(int m, int n) {

    // cuda softmax is hardcoded for now. todo unhardcode
    int HARDCODED_COLS = 10;
    CUPMatrix<float> mlp = CUPMatrix<float>::Random(m, HARDCODED_COLS, -5.f, 5.f);
    EigenMatrix eig = _testEig::fromCUPMatrix<float>(mlp);

    mlp.softmax();
    EigenMatrix mlpCmp = _testEig::fromCUPMatrix<float>(mlp);
    EigenMatrix eigCmp = _testEig::softmax(eig);

    _testEig::cmpMat(eigCmp, mlpCmp);
}
void test_one_hot(int m, int n) {

    n *= 8;
    int outputSize = n * 8; // ensure one_hot produces correct cols
    CUPMatrix<float> mlp = CUPMatrix<float>(m * 8, outputSize);
    EigenMatrix eig = EigenMatrix(m * 8, outputSize);

    CUPMatrix<int> y = CUPMatrix<int>::Random(m * 8, 1, 0, outputSize - 1);
    EigenMatrix yEig = _testEig::fromCUPMatrix<int>(y);
    mlp.oneHot(y, outputSize);
    EigenMatrix mlpCmp = _testEig::fromCUPMatrix<float>(mlp);
    EigenMatrix eigCmp = _testEig::one_hot(yEig, outputSize);

    _testEig::cmpMat(eigCmp, mlpCmp);
}
void test_dup_rows(int m, int n) {
    n *= 8;
    CUPMatrix<float> mlp = CUPMatrix<float>::Random(m, n * 8, 0.f, 0.f);
    EigenMatrix eig = _testEig::fromCUPMatrix<float>(mlp);

    CUPMatrix<float> row = CUPMatrix<float>::Random(n * 8, 1, -5.f, 5.f);
    EigenMatrix eigRow = _testEig::fromCUPMatrix<float>(row);
    mlp.dupRows2(row, mlp.rows);
    EigenMatrix mlpCmp = _testEig::fromCUPMatrix<float>(mlp);
    EigenMatrix eigCmp = _testEig::dup_rows(eig, eigRow);

    _testEig::cmpMat(eigCmp, mlpCmp);

}

void test_positive_mask(int m, int n) {
    n *= 8;
    CUPMatrix<float> mlp = CUPMatrix<float>::Random(m, n * 8, -5.f, 5.f);
    CUPMatrix<float> mlpMask = CUPMatrix<float>::Random(m, n * 8, -5.f, 5.f);
    EigenMatrix eig = _testEig::fromCUPMatrix<float>(mlp);
    EigenMatrix eigMask = _testEig::fromCUPMatrix<float>(mlpMask);

    mlp.positiveMask(mlpMask);
    EigenMatrix mlpCmp = _testEig::fromCUPMatrix<float>(mlp);
    EigenMatrix eigCmp = _testEig::positive_mask(eig, eigMask);

    _testEig::cmpMat(eigCmp, mlpCmp);
}

void test_raii() {
    CUPMatrix<int> a{ 5,5, 1 };
    a = CUPMatrix<int>{ 6,6, 2 }; // expanding
    a = CUPMatrix<int>{ 4,4, 3 }; // shrinking

    CUPMatrix<int> b;
    b = a;
    assert(b.data != a.data); // deep copy (assignment)
    CUPMatrix<int> copyCtor(a);
    assert(copyCtor.data != a.data); // deep copy (copy ctor)
    PODMatrix<int> pod = a.getPod();
    assert(pod.data == a.data); // weakref copy

    a.setView(1, 2);
    CUPMatrix<int> copyView1 = a;
    assert(copyView1.getRowOffset() == 1);
    assert(copyView1.rows == 2);
    assert(copyView1.raiiRows() == 4);

    CUPMatrix<int> copyView2(copyView1);
    assert(copyView2.getRowOffset() == 1);
    assert(copyView2.rows == 2);
    assert(copyView2.raiiRows() == 4);

    copyView2.setView(0, copyView2.raiiRows());
    auto dataPtr = thrust::device_pointer_cast(copyView2.data);
    assert(*dataPtr == 3);
    auto endPtr = thrust::device_pointer_cast(copyView2.end() - 1);
    assert(*endPtr == 3);
}

void test_colwiseSum(int m, int n, CUPMatrix<float>& ones) {
    CUPMatrix<float> b = CUPMatrix<float>::Random(m, n, -100.f, 100.f);
    EigenMatrix bEig = _testEig::fromCUPMatrix(b);
    CUPMatrix<float> bSum;
    float fAlpha = 1.2f;

    bSum.colwiseSumAlpha(b, ones, fAlpha);
    EigenMatrix bSumEig = (bEig.colwise().sum() * fAlpha);

    assert(bSum.rows == 1 && bSum.cols == b.cols);

    EigenMatrix bSumCmp = _testEig::fromCUPMatrix(bSum);
    _testEig::cmpMat(bSumEig, bSumCmp);

}

void testRun() {
    test_raii();
    std::vector<int> in, out;
    CUPMatrix<float> ones;
    in = { 1,2,3,4,5 };
    out = std::vector<int>(2, 0);
    while (nextPermute(in, out)) {
        int m = out[0];
        int n = out[1];
        test_colwiseSum(m, n, ones);
        test_softmax(m, n);
        test_relu(m, n);
        test_one_hot(m, n);
        test_dup_rows(m, n);
        test_positive_mask(m, n);
    }
    in = { 1,2,3,4,5 };
    out = std::vector<int>(3, 0);
    while (nextPermute(in, out)) {
        int m = out[0];
        int n = out[1];
        int k = out[2];
        test_gemm(m, n, k);
    }
    std::cout << "Passed all unit tests\n";
}

int main() {
    enableFpExcept();
    testRun();

    CUPMatrix<float> x = readIdxXubyte<float>("assets.ignored/train-images.idx3-ubyte");
    CUPMatrix<int> y = readIdxXubyte<int>("assets.ignored/train-labels.idx1-ubyte");
    CUPMatrix<float> testX = readIdxXubyte<float>("assets.ignored/t10k-images.idx3-ubyte");
    CUPMatrix<int> testY = readIdxXubyte<int>("assets.ignored/t10k-labels.idx1-ubyte");

    constexpr size_t hiddenSize = 128;
    constexpr int epochs = 110;
    constexpr int batchSize = 128;
    constexpr float lr = 0.01f;

    Time begin = getTime();
    MLP mlp{ x, y, hiddenSize, batchSize, lr, epochs };
    Seconds elapsed = getTime() - begin;
    std::cout << "training time: " << elapsed << "\n";

    std::vector<int> predictions = mlp.predict(testX);
    std::vector<int> testYEig = testY.cpyFromDevice();
    assert(predictions.size() == testYEig.size());

    float acc = 0;
    for (int i = 0; i < predictions.size(); ++i) {
        if (predictions[i] == testYEig[i]) {
            ++acc;
        }
    }
    std::cout << "Test Accuracy: " << acc / float(predictions.size()) << std::endl;

    CublasHandle::free();
    return 0;
}

